#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample illustrates the usage of CUDA events for both GPU timing and
 * overlapping CPU and GPU execution.  Events are inserted into a stream
 * of CUDA calls.  Since CUDA stream calls are asynchronous, the CPU can
 * perform computations while GPU is executing (including DMA memcopies
 * between the host and device).  CPU can query CUDA events to determine
 * whether GPU has completed tasks.
 */
// https://zhuanlan.zhihu.com/p/598109614  CUDA Samplesѧϰ�ʼ�: 0_Sample/asyncAPI
// includes, system
#include <stdio.h>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>  // helper utility functions

__global__ void increment_kernel(int *g_data, int inc_value) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  g_data[idx] = g_data[idx] + inc_value;
}

bool correct_output(int *data, const int n, const int x) {
  for (int i = 0; i < n; i++)
    if (data[i] != x) {
      printf("Error! data[%d] = %d, ref = %d\n", i, data[i], x);
      return false;
    }

  return true;
}

int main(int argc, char *argv[]) {
  int devID;
  hipDeviceProp_t deviceProps;

  printf("[%s] - Starting...\n", argv[0]);

  // This will pick the best possible CUDA capable device  ��������ʵ�cuda�豸����main�����Ŀ�ͷ���ã�����deviceID
  devID = findCudaDevice(argc, (const char **)argv);

  // get device name
  checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
  printf("CUDA device [%s]\n", deviceProps.name);

  int n = 16 * 1024 * 1024;
  int nbytes = n * sizeof(int);
  int value = 26;

  // allocate host memory
  int *a = 0;
  checkCudaErrors(hipHostMalloc((void **)&a, nbytes));
  memset(a, 0, nbytes);

  // allocate device memory
  int *d_a = 0;
  checkCudaErrors(hipMalloc((void **)&d_a, nbytes));
  checkCudaErrors(hipMemset(d_a, 255, nbytes));

  // set kernel launch configuration
  dim3 threads = dim3(512, 1);
  dim3 blocks = dim3(n / threads.x, 1);

  // create cuda event handles
  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);
  sdkResetTimer(&timer);

  checkCudaErrors(hipDeviceSynchronize());
  float gpu_time = 0.0f;

  // asynchronously issue work to the GPU (all to stream 0)
  checkCudaErrors(hipProfilerStart());
  sdkStartTimer(&timer);
  hipEventRecord(start, 0);
  hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, 0);
  increment_kernel<<<blocks, threads, 0, 0>>>(d_a, value);
  hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, 0);
  hipEventRecord(stop, 0);
  sdkStopTimer(&timer);
  checkCudaErrors(hipProfilerStop());

  // have CPU do some work while waiting for stage 1 to finish
  unsigned long int counter = 0;

  while (hipEventQuery(stop) == hipErrorNotReady) {
    counter++;
  }

  checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));

  // print the cpu and gpu times
  printf("time spent executing by the GPU: %.2f\n", gpu_time);
  printf("time spent by CPU in CUDA calls: %.2f\n", sdkGetTimerValue(&timer));
  printf("CPU executed %lu iterations while waiting for GPU to finish\n",
         counter);

  // check the output for correctness
  bool bFinalResults = correct_output(a, n, value);

  // release resources
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));
  checkCudaErrors(hipHostFree(a));
  checkCudaErrors(hipFree(d_a));

  exit(bFinalResults ? EXIT_SUCCESS : EXIT_FAILURE);
}

/*
������ͨ��CUDA event��չʾGPU��CPU+GPU�غ�����µĳ���ʱ��ͨ����CUDA�����в����¼�(Event), 
��CPU�Ͽ��Բ�ѯCUDA Event�ķ�ʽ��ȷ��GPU�ϵĲ����Ƿ��Ѿ�ִ����ɡ�

������չʾ�첽API��ʹ�÷�����ʹ���첽API��GPU�ں˲�����ͨ��cudaEvent������¼GPU��ִ��ʱ�䡣
ʹ�õ��첽�����У� 1)��Host��device�����ڴ棻 2)kernel����������3)device�ϵ�����copy��host�� 
���ͨ���¼�״̬���ķ�ʽ��¼GPU�ϲ�����ʱ�䣻

���ĸ��

�첽���ݴ��䣺hipMemcpyAsync
CUDA Stream: A sequence of CUDA commands
Event: hipEventCreate
Asynchronous Data Transfers, CUDA Streams and Events

���ĺ���
hipEventCreate(&start)�� 
������һ��GPUʱ��������ʱ��������û�ָ����ʱ����ϼ�¼�ġ�
�ú�������ֱ����GPU�ϼ�¼ʱ�䣻�������첽������ʼʱʹ��hipEventCreate(&start)����¼��ʼʱ�䣬
���첽�����������ʱͨ��hipEventCreate(&stop)��¼����ʱ�䣻֮��start��stop�ᱻGPU���ʱ�����
����start/stop֧��ͨ��hipEventQuery����ѯ״̬��

hipEventQuery:ͨ����API��ѯevent�Ƿ�ready��������event�Ƿ�ִ����ȷ��stop�¼��Ƿ�GPU��¼��

�������
�¼���hipEvent_t

�¼���أ�ͨ��hipEventCreate��������ʼ������Event��Event������ΪhipEvent_t��

    // create cuda event handles
 hipEvent_t start, stop;
 checkCudaErrors(hipEventCreate(&start));
 checkCudaErrors(hipEventCreate(&stop));
��Ӧ��event��������� hipEventDestroy

ͬ����
hipDeviceSynchronize: ��������GPU������ʼ֮ǰ���ô˺�����
�Ժ�������Ϊ������ǰ����ֱ��CUDA device ready��
�ڴ��൱�ڵȴ�ֱ��GPU״̬���ã�

�ڴ���أ�
�������ڴ�ʹ�ö��cuda kernel����ÿ���ڴ��ַ����һ��ֵ���ڴ�����host�Ϸ���ã�Ȼ�����ó�ֵΪ255�� 
֮����host��������ͬ��С�����ݣ��ٽ�host�ϳ�ֵΪ255���ڴ��copy��GPU�ϣ�
֮��ʼkernel�Ĳ���������ÿ���ڴ��ӹ̶�ֵ26����ɺ�������copy��host������correct_output�����ּ�顣

��host�Ϸ����ڴ� cudaAllocHost;
���豸�Ϸ����ڴ�&���ó�ֵ��hipMalloc; hipMemset ;
�ڴ�copy��hipMemcpyAsync��ͨ��hipMemcpyHostToDevice��hipMemcpyDeviceToHostָ��copy�ķ���
�ڴ洴��������
cudaAllocHost -> hipHostFree (host�ϴ������ͷţ�
cudaAlloc -> CudaFree ��device�ϴ������ͷţ�
 hipEventRecord(start, 0);
 hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, 0);
 increment_kernel<<<blocks, threads, 0, 0>>>(d_a, value);
 hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, 0);
 hipEventRecord(stop, 0);
���첽����APIִ�����Ժ�ͨ����ѯcuda�¼��±ߴ������ȴ�ִ�е���ɣ�

  while (hipEventQuery(stop) == hipErrorNotReady)   {  counter++;}
ִ��ʱ��ļ�¼

����ͨ���±߷�ʽ����¼GPU��CPU�ϵ�ִ��ʱ��

GPU��ִ��ʱ�䣺 ��hipEventElapsedTime���ص�ʱ�䵥λΪms��

hipEventElapsedTime(&gpu_time, start, stop)
printf("time spent executing by the GPU: %.2f\n", gpu_time);
CPUִ��ʱ���¼�����ص�λҲ��ms

 StopWatchInterface *timer = NULL;
 sdkCreateTimer(&timer);
 sdkResetTimer(&timer);

 sdkStartTimer(&timer);
// ���ֲ���
 sdkStopTimer(&timer);
 printf("time spent by CPU in CUDA calls: %.2f\n", sdkGetTimerValue(&timer));
 
 */