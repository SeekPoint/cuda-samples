#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample illustrates the usage of CUDA events for both GPU timing and
 * overlapping CPU and GPU execution.  Events are inserted into a stream
 * of CUDA calls.  Since CUDA stream calls are asynchronous, the CPU can
 * perform computations while GPU is executing (including DMA memcopies
 * between the host and device).  CPU can query CUDA events to determine
 * whether GPU has completed tasks.
 */
// https://zhuanlan.zhihu.com/p/598109614  CUDA Samplesѧϰ�ʼ�: 0_Sample/asyncAPI
// https://blog.csdn.net/zcy0xy/article/details/84335367 CUDA samplesϵ�� 0.1 asyncAPI
// includes, system
#include <stdio.h>

// includes CUDA Runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>  // helper utility functions


/*
��������˵��ÿ��thread����ô������Щ�����ģ�����������Ĳ���(d_a, value)�Ǵ���ÿ�����̵ģ�Ȼ���ں˺����ڣ�

��������̱�ţ� int idx = blockIdx.x * blockDim.x + threadIdx.x;  blockDim.x ��һ��block�����ж�ø����̡�

Ȼ����̾Ϳ���֪���ָ��ҵİ�Ķ���ֵ��������ˣ�ĬĬ��ȥ���ҵĻ�������ˣ�

g_data[idx] = g_data[idx] + inc_value;

��ס���˺����������Ǵ��ݵ��ڴ��׵�ַ����ʱ��ֱ�Ӹ����׵�ַ+ƫ�ƾͿ��Եõ��ң�ĳ���̣߳������䵽�İ�Ķ���ֵ��ˡ�
*/
__global__ void increment_kernel(int *g_data, int inc_value) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  g_data[idx] = g_data[idx] + inc_value;
}

bool correct_output(int *data, const int n, const int x) {
  for (int i = 0; i < n; i++)
    if (data[i] != x) {
      printf("Error! data[%d] = %d, ref = %d\n", i, data[i], x);
      return false;
    }

  return true;
}

//���� int main(int argc, char *argv[])�Լ�c++��֪ʶ,����֪��argc��ָ����������ĸ���,
//����㲻����Ļ�,argc=1,argv="����õ�����Ŀ�ִ���ļ�·��";
int main(int argc, char *argv[]) {
  int devID;
  hipDeviceProp_t deviceProps;

  printf("[%s] - Starting...\n", argv[0]);

  // This will pick the best possible CUDA capable device  
  //��������ʵ�cuda�豸����main�����Ŀ�ͷ���ã�����deviceID
  // ����㲻������,�������ҵ����Gflops/s���Կ�,Ҳ���Ǹ����������ٶ�����;
  devID = findCudaDevice(argc, (const char **)argv);

  //hipGetDeviceProperties(&deviceProps, devID) ����˼��,�����Կ���ID,�õ�����Կ�������;
  // checkCudaErrors() �ܶ�cuda�Դ��ĺ�������״̬����ֵ��,
  // ���ִ�д���Ļ�,�ͷ��ش���ı��,���checkCudaErrors()ר���������ݴ���ı����ʾ������Ϣ,���û�д���,��ͨ����,�����ж�������;
  // get device name
  checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
  printf("CUDA device [%s]\n", deviceProps.name);

  int n = 16 * 1024 * 1024;
  int nbytes = n * sizeof(int);
  int value = 26;

  // allocate host memory
  int *a = 0;
  checkCudaErrors(hipHostMalloc((void **)&a, nbytes));
  memset(a, 0, nbytes);

  // allocate device memory
  int *d_a = 0;
  checkCudaErrors(hipMalloc((void **)&d_a, nbytes));
  checkCudaErrors(hipMemset(d_a, 255, nbytes));

  // set kernel launch configuration
  dim3 threads = dim3(512, 1);
  dim3 blocks = dim3(n / threads.x, 1);

  /*������ʱ����
�������ǵĴ������֮ǰ,��ס2����ʱ�ĺ���,һ����cpu��ʱ����,
���������sdkStartTimer(&timer) �Լ� sdkStopTimer(&timer) ֮��ĳ��������ʱ��,
����������������ʲôʱ��ִ����? ���������������е������ʱ��, Ҳ����cpu�õ�������Ŀ���Ȩ��ʱ��.

������˵��ôһ��ϻ�����Ϊ��δ��벢������������ǰ��c++����һ��,��һ��ִ�����˲Ž�����һ��,.
��������趨,������ó�����ͳ��c++һ��,��ִ������<GPU������������>,�Ż�ִ����һ��sdkStopTimer(&timer)������ʱ,�����"ͬ��ִ��";
������Ҳ�������Կ�ִ��<GPU������������>,���ͬʱ���cpuֱ��ִ�������sdkStopTimer(&timer),�����"�첽ִ��".
���ϵĺ����ֲ���ʮ��׼ȷ,�����ڴ��������������"ͬ��"��"�첽".

    */

  // ����
  // create cuda event handles  
  hipEvent_t start, stop;

  // ����
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);
  sdkResetTimer(&timer);

  checkCudaErrors(hipDeviceSynchronize());
  float gpu_time = 0.0f;

  // asynchronously issue work to the GPU (all to stream 0)
  checkCudaErrors(hipProfilerStart());
  sdkStartTimer(&timer); // cpu��ʼ��ʱ

  //<GPU������������>


  /*
  ����stream, ������ʱ��ô���,һ��stream���൱��һ��������main�����Ĵ���,�������г���demo,���Դ��ն�,���������./demo,
  �س�,��ô��򿪼����ն˾Ϳ��Զ����м�������. ��cuda��������������һ�ݴ�����ִ�кü�������������������,һ��stream����һ��main��������,
  ����Կ���cuda���������ڲ�ͬstream�Ĵ���,��ֻҪ��ס���ǵı����ǲ�ͬmain����,�������,����cuda���������ǿ���������,��һ������˲�ִ����һ��.

    ����ļ�ʱ����,��Ҫ���stream�ı��,��Ϊ���������ڲ�ͬstream�ļ�ʱ����,ֻ��ָ����streamִ�е����������Ż��һ��ʱ��,
    �����ĳ����ߵ�������������������,��������������(����cpu������)Ҳ����.

    ������ῴ����2����ʱ����λ�ڴ�����ͬ����λ��,Ȼ���õ���ʱ��ȴ����ͬ,ԭ��ܼ�,��Ϊ���Ǹ���������Ϊ2�������ĳ����ʱ����.

    ��ͼ������ https://blog.csdn.net/zcy0xy/article/details/84335367

    ���Կ�����������ʱ�������ڶԲ�ͬ�Ĵ�����м�ʱ������õ���ͬ��ʱ����Ҳ����������Ȼ���ˡ�
    */


    //��gpu�����streamִ�е�����ʱ,���һ�����ʱ���
    //�����0ָ����stream�ı��,0��stream
  hipEventRecord(start, 0);  // Gpu��ʼ��ʱ

  // ����
  hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, 0);

  /*
  �ġ��˺����ĵ���
        �� CUDA �У�Ҫִ��һ���˺�����ʹ�����µ��﷨��
        ��������<<<block ��Ŀ, thread ��Ŀ, shared memory ��С�� stream���>>>(����...);
        block�Ǻܶ��thread�ļ��ϣ�����˼��block���飬Ҳ���ǽ��̿飻
        thread�ǽ��̣�
        <<<m, n, 0, 0>>>��ĵ�һ���������ܹ�׼������m��block���ڶ�����ÿ��block����n�����̣������ܹ�����m*n�����̡�
        �����������������ڴ��С��������Ϊ0��
        ���ĸ�����������ָ���ĸ�stream�ˣ�ָ������������������ĸ�stream��
  */
  increment_kernel<<<blocks, threads, 0, 0>>>(d_a, value);
  hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, 0);
  hipEventRecord(stop, 0); // Gpu������ʱ

  sdkStopTimer(&timer); // cpu������ʱ
  checkCudaErrors(hipProfilerStop());

  // have CPU do some work while waiting for stage 1 to finish
  unsigned long int counter = 0;

  //ѭ����������Ǽ�鵽stop��ǣ� ֻ��stream0�ߵ�stop�Ż��鵽
  while (hipEventQuery(stop) == hipErrorNotReady) {  
    counter++;
  }

  checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));

  // print the cpu and gpu times
  printf("time spent executing by the GPU: %.2f\n", gpu_time);
  printf("time spent by CPU in CUDA calls: %.2f\n", sdkGetTimerValue(&timer));
  printf("CPU executed %lu iterations while waiting for GPU to finish\n",
         counter);

  // check the output for correctness
  bool bFinalResults = correct_output(a, n, value);

  // release resources
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));
  checkCudaErrors(hipHostFree(a));
  checkCudaErrors(hipFree(d_a));

  exit(bFinalResults ? EXIT_SUCCESS : EXIT_FAILURE);
}

/*
������ͨ��CUDA event��չʾGPU��CPU+GPU�غ�����µĳ���ʱ��ͨ����CUDA�����в����¼�(Event), 
��CPU�Ͽ��Բ�ѯCUDA Event�ķ�ʽ��ȷ��GPU�ϵĲ����Ƿ��Ѿ�ִ����ɡ�

������չʾ�첽API��ʹ�÷�����ʹ���첽API��GPU�ں˲�����ͨ��cudaEvent������¼GPU��ִ��ʱ�䡣
ʹ�õ��첽�����У� 1)��Host��device�����ڴ棻 2)kernel����������3)device�ϵ�����copy��host�� 
���ͨ���¼�״̬���ķ�ʽ��¼GPU�ϲ�����ʱ�䣻

���ĸ��

�첽���ݴ��䣺hipMemcpyAsync
CUDA Stream: A sequence of CUDA commands
Event: hipEventCreate
Asynchronous Data Transfers, CUDA Streams and Events

���ĺ���
hipEventCreate(&start)�� 
������һ��GPUʱ��������ʱ��������û�ָ����ʱ����ϼ�¼�ġ�
�ú�������ֱ����GPU�ϼ�¼ʱ�䣻�������첽������ʼʱʹ��hipEventCreate(&start)����¼��ʼʱ�䣬
���첽�����������ʱͨ��hipEventCreate(&stop)��¼����ʱ�䣻֮��start��stop�ᱻGPU���ʱ�����
����start/stop֧��ͨ��hipEventQuery����ѯ״̬��

hipEventQuery:ͨ����API��ѯevent�Ƿ�ready��������event�Ƿ�ִ����ȷ��stop�¼��Ƿ�GPU��¼��

�������
�¼���hipEvent_t

�¼���أ�ͨ��hipEventCreate��������ʼ������Event��Event������ΪhipEvent_t��

    // create cuda event handles
 hipEvent_t start, stop;
 checkCudaErrors(hipEventCreate(&start));
 checkCudaErrors(hipEventCreate(&stop));
��Ӧ��event��������� hipEventDestroy

ͬ����
hipDeviceSynchronize: ��������GPU������ʼ֮ǰ���ô˺�����
�Ժ�������Ϊ������ǰ����ֱ��CUDA device ready��
�ڴ��൱�ڵȴ�ֱ��GPU״̬���ã�

�ڴ���أ�
�������ڴ�ʹ�ö��cuda kernel����ÿ���ڴ��ַ����һ��ֵ���ڴ�����host�Ϸ���ã�Ȼ�����ó�ֵΪ255�� 
֮����host��������ͬ��С�����ݣ��ٽ�host�ϳ�ֵΪ255���ڴ��copy��GPU�ϣ�
֮��ʼkernel�Ĳ���������ÿ���ڴ��ӹ̶�ֵ26����ɺ�������copy��host������correct_output�����ּ�顣

��host�Ϸ����ڴ� cudaAllocHost;
���豸�Ϸ����ڴ�&���ó�ֵ��hipMalloc; hipMemset ;
�ڴ�copy��hipMemcpyAsync��ͨ��hipMemcpyHostToDevice��hipMemcpyDeviceToHostָ��copy�ķ���
�ڴ洴��������
cudaAllocHost -> hipHostFree (host�ϴ������ͷţ�
cudaAlloc -> CudaFree ��device�ϴ������ͷţ�
 hipEventRecord(start, 0);
 hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, 0);
 increment_kernel<<<blocks, threads, 0, 0>>>(d_a, value);
 hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, 0);
 hipEventRecord(stop, 0);
���첽����APIִ�����Ժ�ͨ����ѯcuda�¼��±ߴ������ȴ�ִ�е���ɣ�

  while (hipEventQuery(stop) == hipErrorNotReady)   {  counter++;}
ִ��ʱ��ļ�¼

����ͨ���±߷�ʽ����¼GPU��CPU�ϵ�ִ��ʱ��

GPU��ִ��ʱ�䣺 ��hipEventElapsedTime���ص�ʱ�䵥λΪms��

hipEventElapsedTime(&gpu_time, start, stop)
printf("time spent executing by the GPU: %.2f\n", gpu_time);
CPUִ��ʱ���¼�����ص�λҲ��ms

 StopWatchInterface *timer = NULL;
 sdkCreateTimer(&timer);
 sdkResetTimer(&timer);

 sdkStartTimer(&timer);
// ���ֲ���
 sdkStopTimer(&timer);
 printf("time spent by CPU in CUDA calls: %.2f\n", sdkGetTimerValue(&timer));
 
 */


/*
�塢�ܽ�
����gpu����һ�㰴���岽�ߣ�

����һ���ڴ�ռ�A��hipMalloc����hipHostMalloc���ڶ����������ٵĿռ������cpu��gpuͬʱ���ʵ�����һ��ֻ����gpu���ʣ����ǵ�һ��Ҫ��ࣩܶ
����Ҫ��������ݿ�����A
ִ������
�������������ص�cpu
жĥɱ¿�����Ӳ��ţ��ͷſ��ٵĿռ�A��hipHostFree(A)��hipFree(A)��
�����������

[/root/cuda-workspace/asyncAPI/Release/asyncAPI] - Starting...
GPU Device 0: "GeForce GTX 1080 Ti" with compute capability 6.1

CUDA device [GeForce GTX 1080 Ti]
time spent executing by the GPU: 11.05
time spent by CPU in CUDA calls: 0.03
CPU executed 49566 iterations while waiting for GPU to finish
����ʱ�䲻һ�������˵���ˡ�

��������һ�������Ǹ�while�������֪��cpu��ѯ��stream0�ߵ���<stop��ǵ�>���Ż��˳�ѭ����

����ǵ�һ�����̣����������Ҫ�ѣ��漰�˲��ñ�Ӧ�����к��ڵ�֪ʶ�㣬����nvidia�ٷ��������samples������ѭ�򽥽����Ѷȡ�

���ǵ�һƪ��ϣ���Լ���д��ȥ���ص��ס�

��11.23��ע���Ⲣ���ǵ�һ�����̣�ֻ�����ǰ�����ĸ����ģ��������asyncAPI�ǵ�һ���������������������д�򵥵����̡���


��������������������������������
��Ȩ����������ΪCSDN������zcy0xy����ԭ�����£���ѭCC 4.0 BY-SA��ȨЭ�飬ת���븽��ԭ�ĳ������Ӽ���������
ԭ�����ӣ�https://blog.csdn.net/zcy0xy/article/details/84335367
*/