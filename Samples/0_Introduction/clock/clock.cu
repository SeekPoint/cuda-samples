#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This example shows how to use the clock function to measure the performance
 * of block of threads of a kernel accurately. Blocks are executed in parallel
 * and out of order. Since there's no synchronization mechanism between blocks,
 * we measure the clock once for each block. The clock samples are written to
 * device memory.
 */

// System includes
#include <assert.h>
#include <stdint.h>
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
/*
* 
���������Ҫ��չʾ�����趨��block�����ӣ�ʱ�䲢����Խ��blockԽ�죬��������һ�������ƣ���������ԵĴ���ӵ�£�latencyԽ��Խ�������ء�
��̬�����ڴ�
�����д�˺���ʱ��֪�������ڴ�Ĵ�С������ʹ�ö�̬�����ڴ棬�ڵ��ú���ʱ���趨���С������Ϊ�ں˺����ж��壺

extern __shared__ float shared[];

�ڵ��ú˺���ʱ��

timedReduction<<<NUM_BLOCKS, NUM_THREADS,  �����ڴ��С>>>(dinput, doutput, dtimer);
�˺�������

*/
// This kernel computes a standard parallel reduction and evaluates the
// time it takes to do that for each block. The timing results are stored
// in device memory.
__global__ static void timedReduction(const float *input, float *output,
                                      clock_t *timer) {
  // __shared__ float shared[2 * blockDim.x];
  extern __shared__ float shared[];
  //����ʱ��֪��shared memory�Ĵ�С���ڵ���ʱ<<<>>>�е�������������

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;

  if (tid == 0) timer[bid] = clock(); //ÿ��block�еĵ�0��thread��ʱ��ʼ

  // Copy input.
    //���Ƶ�block�Ĺ����ڴ�
    //ÿ��block��256��thread��input����0-255,256-511
  shared[tid] = input[tid]; //block���̺߳� 
  shared[tid + blockDim.x] = input[tid + blockDim.x]; //block���̺߳�+256

  // Perform reduction to find minimum.
  for (int d = blockDim.x; d > 0; d /= 2) {
    __syncthreads();  //�������ȵ�ͬһ��block�ڵ�thread���ߵ�������Ż����

    if (tid < d) {
      float f0 = shared[tid];
      float f1 = shared[tid + d];

      if (f1 < f0) {
        shared[tid] = f1;
      }
    }
  }

  // Write result.
  //ȡÿ��block��0��thread�Ľ����Ϊ���block�����ս��
  if (tid == 0) output[bid] = shared[0];

  __syncthreads();  //ͬһ��block�ڵ�thread���ߵ�������ż���������׼ȷ��ʱ

  if (tid == 0) timer[bid + gridDim.x] = clock();
}

#define NUM_BLOCKS 64
#define NUM_THREADS 256

// It's interesting to change the number of blocks and the number of threads to
// understand how to keep the hardware busy.
//
// Here are some numbers I get on my G80:
//    blocks - clocks
//    1 - 3096
//    8 - 3232
//    16 - 3364
//    32 - 4615
//    64 - 9981
//
// With less than 16 blocks some of the multiprocessors of the device are idle.
// With more than 16 you are using all the multiprocessors, but there's only one
// block per multiprocessor and that doesn't allow you to hide the latency of
// the memory. With more than 32 the speed scales linearly.

// Start the main CUDA Sample here
int main(int argc, char **argv) {
  printf("CUDA Clock sample\n");

  // This will pick the best possible CUDA capable device
  int dev = findCudaDevice(argc, (const char **)argv);

  float *dinput = NULL;
  float *doutput = NULL;
  clock_t *dtimer = NULL;

  clock_t timer[NUM_BLOCKS * 2];
  float input[NUM_THREADS * 2];

  for (int i = 0; i < NUM_THREADS * 2; i++) {
    input[i] = (float)i;
  }

  checkCudaErrors(
      hipMalloc((void **)&dinput, sizeof(float) * NUM_THREADS * 2));
  checkCudaErrors(hipMalloc((void **)&doutput, sizeof(float) * NUM_BLOCKS));
  checkCudaErrors(
      hipMalloc((void **)&dtimer, sizeof(clock_t) * NUM_BLOCKS * 2));

  checkCudaErrors(hipMemcpy(dinput, input, sizeof(float) * NUM_THREADS * 2,
                             hipMemcpyHostToDevice));

  timedReduction<<<NUM_BLOCKS, NUM_THREADS, sizeof(float) * 2 * NUM_THREADS>>>(
      dinput, doutput, dtimer);

  checkCudaErrors(hipMemcpy(timer, dtimer, sizeof(clock_t) * NUM_BLOCKS * 2,
                             hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(dinput));
  checkCudaErrors(hipFree(doutput));
  checkCudaErrors(hipFree(dtimer));

  long double avgElapsedClocks = 0;

  for (int i = 0; i < NUM_BLOCKS; i++) {
    avgElapsedClocks += (long double)(timer[i + NUM_BLOCKS] - timer[i]);
  }

  avgElapsedClocks = avgElapsedClocks / NUM_BLOCKS;
  printf("Average clocks/block = %Lf\n", avgElapsedClocks);

  return EXIT_SUCCESS;
}

/*
����������
����ܼ򵥣����������н��
// block:64 35237
// block:1 26824
// block:8 26823
���Կ���block�����ӱ�Ӧ�ò���Ӱ��ʱ��ģ��������ӵ�64ʱ�����Կ�ʼ�����ˡ�Դ���Ӣ��ע���ᵽ����G80�Կ�������ʱ������32��block��ʱ�����block�������������ˡ�����û�в��Ը����block��Ŀ�ˡ�
��������������������������������
https://blog.csdn.net/zcy0xy/article/details/84502030  CUDA samplesϵ�� 0.5 clock
*/