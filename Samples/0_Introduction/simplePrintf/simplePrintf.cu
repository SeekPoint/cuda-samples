#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#ifndef MAX
#define MAX(a, b) (a > b ? a : b)
#endif

__global__ void testKernel(int val) {
  printf("[%d, %d]:\t\tValue is:%d\n", blockIdx.y * gridDim.x + blockIdx.x,
         threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
             threadIdx.x,
         val);
}

int main(int argc, char **argv) {
  int devID;
  hipDeviceProp_t props;

  // This will pick the best possible CUDA capable device
  devID = findCudaDevice(argc, (const char **)argv);

  // Get GPU information
  checkCudaErrors(hipGetDevice(&devID));
  checkCudaErrors(hipGetDeviceProperties(&props, devID));
  printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name,
         props.major, props.minor);

  printf("printf() is called. Output:\n\n");

  // Kernel configuration, where a two-dimensional grid and
  // three-dimensional blocks are configured.
  dim3 dimGrid(2, 2);
  dim3 dimBlock(2, 2, 2);
  testKernel<<<dimGrid, dimBlock>>>(10);
  hipDeviceSynchronize();

  return EXIT_SUCCESS;
}

/*
GPU Device 0: "GeForce GTX 1070" with compute capability 6.1

Device 0 : "GeForce GTX 1070" with Compute 6.1 capability
[2, 0] : Value is : 10
[2, 1] : Value is : 10
[2, 2] : Value is : 10
[2, 3] : Value is : 10
[2, 4] : Value is : 10
[2, 5] : Value is : 10
[2, 6] : Value is : 10
[2, 7] : Value is : 10
[3, 0] : Value is : 10
[3, 1] : Value is : 10
[3, 2] : Value is : 10
[3, 3] : Value is : 10
[3, 4] : Value is : 10
[3, 5] : Value is : 10
[3, 6] : Value is : 10
[3, 7] : Value is : 10
[1, 0] : Value is : 10
[1, 1] : Value is : 10
[1, 2] : Value is : 10
[1, 3] : Value is : 10
[1, 4] : Value is : 10
[1, 5] : Value is : 10
[1, 6] : Value is : 10
[1, 7] : Value is : 10
[0, 0] : Value is : 10
[0, 1] : Value is : 10
[0, 2] : Value is : 10
[0, 3] : Value is : 10
[0, 4] : Value is : 10
[0, 5] : Value is : 10
[0, 6] : Value is : 10
[0, 7] : Value is : 10
*/