#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
// https ://blog.csdn.net/zcy0xy/article/details/84424182  CUDA samplesϵ�� 0.2 simpleAssert

#ifdef _WIN32
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#else
#include <sys/utsname.h>
#endif

/*
��������У��и��ṹutsname��������Ի�ȡ��Щ��Ϣ��

struct utsname
  { char sysname[_UTSNAME_SYSNAME_LENGTH];//��ǰ����ϵͳ��
   char nodename[_UTSNAME_NODENAME_LENGTH];//�����ϵ�����
   char release[_UTSNAME_RELEASE_LENGTH];//��ǰ��������
   char version[_UTSNAME_VERSION_LENGTH];//��ǰ�����汾
   char machine[_UTSNAME_MACHINE_LENGTH];//��ǰӲ����ϵ����

*/


// Includes, system
#include <stdio.h>
#include <cassert>

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>  // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>  // helper functions for CUDA error check

const char *sampleName = "simpleAssert";

////////////////////////////////////////////////////////////////////////////////
// Auto-Verification Code
bool testResult = true;

////////////////////////////////////////////////////////////////////////////////
// Kernels
////////////////////////////////////////////////////////////////////////////////
//! Tests assert function.
//! Thread whose id > N will print assertion failed error message.
////////////////////////////////////////////////////////////////////////////////
/*
���Ǻ˺����������60��������̺߳�,�����̺߳�<60�󡣿��ٵ��߳���2*32=64���̺߳Ŵ�0��ʼ������60,61,62,63��4���̺߳Żᱻ���Դ���
*/
__global__ void testKernel(int N) {
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
  assert(gtid < N);
}

////////////////////////////////////////////////////////////////////////////////
// Declaration, forward
void runTest(int argc, char **argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  printf("%s starting...\n", sampleName);

  runTest(argc, argv);

  printf("%s completed, returned %s\n", sampleName,
         testResult ? "OK" : "ERROR!");
  exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

void runTest(int argc, char **argv) {
  int Nblocks = 2;
  int Nthreads = 32;
  hipError_t error;

#ifndef _WIN32
  utsname OS_System_Type;
  uname(&OS_System_Type);

  printf("OS_System_Type.release = %s\n", OS_System_Type.release);

  if (!strcasecmp(OS_System_Type.sysname, "Darwin")) {
    printf("simpleAssert is not current supported on Mac OSX\n\n");
    exit(EXIT_SUCCESS);
  } else {
    printf("OS Info: <%s>\n\n", OS_System_Type.version);
  }

#endif

  // This will pick the best possible CUDA capable device
  findCudaDevice(argc, (const char **)argv);

  // Kernel configuration, where a one-dimensional
  // grid and one-dimensional blocks are configured.
  // dim3 ������ͣ�����˺�����block������thread��������������������͡�������1,2,3ά�ģ�������һά�ġ�
  // ʵ���ϣ������ֻдһά�Ļ����������������dim3 dimGrid(Nblocks)�� ���ῴ����dim3 dimGrid(Nblocks��1��1)������ܹ�����ά�ġ�
  dim3 dimGrid(Nblocks);
  dim3 dimBlock(Nthreads);

  printf("Launch kernel to generate assertion failures\n");
  testKernel<<<dimGrid, dimBlock>>>(60);

  // Synchronize (flushes assert output).
  printf("\n-- Begin assert output\n\n");

  /*
  �ȿ���䡣hipDeviceSynchronize() ��������ǰ�����ִ�У�ֱ���������񶼴�����ϣ�
  Ҳ����˵�������ߵ������ȴ���仰֮ǰ�����д���ȫ��ִ������ˣ����е�stream��ִ������ˣ�
  ����stream����һƪ0.1�Ѿ�������ϸֱ�۵Ľ��͡�

  Ҳ����stream�󶨵������ȴ�������hipStreamSynchronize(streamID)����һ��������cuda��ID��
  ��ֻ������Щcuda��ID���ڲ�����ָ��ID����Щcuda���̣�������Щ��ID���ȵ����̣������첽ִ�еġ�
    */
  error = hipDeviceSynchronize();
  printf("\n-- End assert output\n\n");

  // Check for errors and failed asserts in asynchronous kernel launch.
  /*
  �����и�hipErrorAssert���Ǹ�û�ж���ģ�Ӧ����ר�����ʣ������cuda�ٷ��̳����ҵ���
  �����https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html
  */
  if (error == hipErrorAssert) {
    printf(
        "Device assert failed as expected, "
        "CUDA error message is: %s\n\n",
        hipGetErrorString(error));
  }

  testResult = error == hipErrorAssert;
}
